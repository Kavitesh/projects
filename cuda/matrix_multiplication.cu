#include <iostream>
#include <hip/hip_runtime.h>

#define N 3  // Matrix size N x N

// CUDA Kernel for matrix multiplication
__global__ void matrixMul(int *A, int *B, int *C, int n) {
    int row = threadIdx.y;
    int col = threadIdx.x;
    int sum = 0;
    for (int k = 0; k < n; k++) {
        sum += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = sum;
}

int main() {
    int h_A[N*N], h_B[N*N], h_C[N*N];  
    int *d_A, *d_B, *d_C;

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        h_A[i] = i + 1;
        h_B[i] = (i + 1) * 2;
    }

    // Allocate memory on GPU
    hipMalloc(&d_A, N * N * sizeof(int));
    hipMalloc(&d_B, N * N * sizeof(int));
    hipMalloc(&d_C, N * N * sizeof(int));

    // Copy matrices to device
    hipMemcpy(d_A, h_A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with a 2D block of threads
    dim3 threadsPerBlock(N, N);
    matrixMul<<<1, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Matrix C (Result):\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << "\n";
    }

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
