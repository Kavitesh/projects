#include <iostream>
#include <hip/hip_runtime.h>

#define N 4      // Number of data points
#define LR 0.01  // Learning rate
#define EPOCHS 100

// CUDA kernel for forward pass: Compute y_pred = w*x + b
__global__ void forwardPass(float *x, float *w, float *b, float *y_pred, int n) {
    int i = threadIdx.x;
    if (i < n) {
        y_pred[i] = (*w) * x[i] + (*b);
    }
}

// CUDA kernel for calculating gradients
__global__ void computeGradients(float *x, float *y, float *y_pred, float *w_grad, float *b_grad, int n) {
    int i = threadIdx.x;
    if (i < n) {
        atomicAdd(w_grad, 2 * x[i] * (y_pred[i] - y[i]) / n);
        atomicAdd(b_grad, 2 * (y_pred[i] - y[i]) / n);
    }
}

// CUDA kernel for updating weights using gradient descent
__global__ void updateWeights(float *w, float *b, float *w_grad, float *b_grad, float lr) {
    *w -= lr * (*w_grad);
    *b -= lr * (*b_grad);
}

int main() {
    float h_x[N] = {1, 2, 3, 4};   // Inputs
    float h_y[N] = {2, 4, 6, 8};   // Targets
    float h_w = 0.0, h_b = 0.0;    // Model parameters (w, b)
    
    float *d_x, *d_y, *d_w, *d_b, *d_y_pred, *d_w_grad, *d_b_grad;

    // Allocate memory on GPU
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_w, sizeof(float));
    hipMalloc(&d_b, sizeof(float));
    hipMalloc(&d_y_pred, N * sizeof(float));
    hipMalloc(&d_w_grad, sizeof(float));
    hipMalloc(&d_b_grad, sizeof(float));

    // Copy data to GPU
    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, &h_w, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(float), hipMemcpyHostToDevice);

    // Training loop
    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        float h_w_grad = 0, h_b_grad = 0;
        hipMemcpy(d_w_grad, &h_w_grad, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b_grad, &h_b_grad, sizeof(float), hipMemcpyHostToDevice);

        forwardPass<<<1, N>>>(d_x, d_w, d_b, d_y_pred, N);
        computeGradients<<<1, N>>>(d_x, d_y, d_y_pred, d_w_grad, d_b_grad, N);
        updateWeights<<<1, 1>>>(d_w, d_b, d_w_grad, d_b_grad, LR);

        hipMemcpy(&h_w, d_w, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&h_b, d_b, sizeof(float), hipMemcpyDeviceToHost);

        if (epoch % 20 == 0) {
            std::cout << "Epoch " << epoch << " - w: " << h_w << ", b: " << h_b << std::endl;
        }
    }

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_w);
    hipFree(d_b);
    hipFree(d_y_pred);
    hipFree(d_w_grad);
    hipFree(d_b_grad);

    std::cout << "Final Model: y = " << h_w << "x + " << h_b << std::endl;
    return 0;
}
