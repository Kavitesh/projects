#include <iostream>
#include <hip/hip_runtime.h>

#define N 10  // Size of vectors

// CUDA Kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int h_a[N], h_b[N], h_c[N];  // Host arrays
    int *d_a, *d_b, *d_c;        // Device arrays

    // Allocate memory on GPU
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Copy data from host to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with N threads
    vectorAdd<<<1, N>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Vector Addition Result:\n";
    for (int i = 0; i < N; i++) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << "\n";
    }

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
